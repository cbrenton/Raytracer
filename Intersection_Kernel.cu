#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include "vector.h"

// includes, project
#include <cutil.h>

#include "Intersection_K.h"

__device__ float dot_product(float *v1, float *v2)
{
    return v1[0] * v2[0] + v1[1] * v2[1] + v1[2] * v2[2];
}

__device__ float hit(ray_dev_t ray, float *t, sphere_dev_t *sp)
{
   float oMinusC[3];
   oMinusC[0] = ray.point[0] - sp->point[0];
   oMinusC[1] = ray.point[1] - sp->point[1];
   oMinusC[2] = ray.point[2] - sp->point[2];   
  
   float _b = dot_product(ray.dir,oMinusC);
   float _c = dot_product(oMinusC,oMinusC) - (sp->rad * sp->rad);
   float det = _b * _b - _c;

   if (det < 0)
   {
      return -1;
   }

   float t0 = -_b - (float)sqrt(det);
   float t1 = -_b + (float)sqrt(det);

   if (t0 >= 0 && t1 >= 0.0)
   {
      if (t0 < t1)
      {
         return t0;
      }
      else if (t1 < t0)
      {
         return t1;
      }
   }
   else if (t0 >= 0)
   {
      return t0;
   }
   else if (t1 >= 0)
   {
      return t1;
   }

   return -1;
}

__global__ void Intersection_Test(ray_dev_t *rays, sphere_dev_t *spheres, intersection_t *intersects, int size, int width, int height)
{
   float t;
   bool hitFound = false;
   float curDepth = -1.0;
   int x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
   int y = blockIdx.y * BLOCK_SIZE + threadIdx.y;

   if(x > width || y > height)
       return;

   ray_dev_t *ray = rays + x * height + y;   
   intersection_t *result = intersects + x * height + y;

   result->index = -1;

   for (int i = 0; i < size; i++)
   {
      sphere_dev_t *curObject = &spheres[i];
      t = -1.0;
      t = hit(*ray, &t, curObject);

      if (t >= 0)
      {
         if (t >= 0 && (!hitFound || (hitFound && t < curDepth)))
         {
            curDepth = t;
            result->index = i;
         }
      }
      hitFound |= (t >= 0 && curDepth > 0.0);
   }
   
   //Add intersection
   result->t = curDepth;
}

