#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include "vector.h"

// includes, project
#include <cutil.h>

#include "Intersection_K.h"

__device__ float dot_product(float *v1, float *v2)
{
<<<<<<< HEAD
   return v1[0] * v2[0] + v1[1] * v2[1] + v1[2] * v2[2];
}

__device__ float det3(float **v1)
{
   return v1[0][0] * v1[1][1] * v1[2][2] + v1[0][1] * v1[1][2] * v1[2][0] + v1[0][2] * v1[1][0] * v1[2][1] -
      v1[0][2] * v1[1][1] * v1[2][0] - v1[0][0] * v1[1][2] * v1[2][1] - v1[0][1] * v1[1][0] * v1[2][2];
   ;
}

__device__ float hit(ray_dev_t ray, triangle_dev_t *tri);
{
   float result = -1;
   float bBeta, bGamma, bT;
   float detA;

   float A[3][3];
   A[0][0] = location.x()-corner2.x();
   A[0][1] = location.x()-corner3.x();
   A[0][2] = ray.dir.x();
   A[1][0] = location.y()-corner2.y();
   A[1][1] = location.y()-corner3.y();
   A[1][2] = ray.dir.y();
   A[2][0] = location.z()-corner2.z();
   A[2][1] = location.z()-corner3.z();
   A[2][2] = ray.dir.z();

   detA = det3(A);

   float baryT[3][3];
   baryT[0][0] = location.x()-corner2.x();
   baryT[0][1] = location.x()-corner3.x();
   baryT[0][2] = location.x()-ray.point.x();
   baryT[1][0] = location.y()-corner2.y();
   baryT[1][1] = location.y()-corner3.y();
   baryT[1][2] = location.y()-ray.point.y();
   baryT[2][0] = location.z()-corner2.z();
   baryT[2][1] = location.z()-corner3.z();
   baryT[2][2] = location.z()-ray.point.z();

   bT = det3(baryT) / detA;
   if (bT < 0)
   {
      result = 0;
   }
   else
   {
      float baryGamma[3][3];

      baryGamma[0][0] = location.x()-corner2.x();
      baryGamma[0][1] = location.x()-ray.point.x();
      baryGamma[0][2] = ray.dir.x();
      baryGamma[1][0] = location.y()-corner2.y();
      baryGamma[1][1] = location.y()-ray.point.y();
      baryGamma[1][2] = ray.dir.y();
      baryGamma[2][0] = location.z()-corner2.z();
      baryGamma[2][2] = location.z()-ray.point.z();
      baryGamma[2][3] = ray.dir.z();

      bGamma = det3(baryGamma) / detA;
      if (bGamma < 0 || bGamma > 1)
      {
         result = 0;
      }
      else
      {
         float baryBeta[3][3];

         baryBeta[0][0] = location.x()-ray.point.x();
         baryBeta[0][1] = location.x()-corner3.x();
         baryBeta[0][2] = ray.dir.x();
         baryBeta[1][0] = location.y()-ray.point.y();
         baryBeta[1][1] = location.y()-corner3.y();
         baryBeta[1][2] = ray.dir.y();
         baryBeta[2][0] = location.z()-ray.point.z();
         baryBeta[2][1] = location.z()-corner3.z();
         baryBeta[2][2] = ray.dir.z();

         bBeta = det3(baryBeta) / detA;
         if (bBeta < 0 || bBeta > 1 - bGamma)
         {
            result = 0;
         }
      }
   }

   if (result != 0)
   {
      result = bT;
   }
   return (result);
}

__device__ float hit(ray_dev_t ray, sphere_dev_t *sp)
=======
    return v1[0] * v2[0] + v1[1] * v2[1] + v1[2] * v2[2];
}

__device__ float hit(ray_dev_t ray, float *t, sphere_dev_t *sp)
>>>>>>> 4fabfe5fce695248b5da2724ded1529aa2c51cdb
{
   float oMinusC[3];
   oMinusC[0] = ray.point[0] - sp->point[0];
   oMinusC[1] = ray.point[1] - sp->point[1];
<<<<<<< HEAD
   oMinusC[2] = ray.point[2] - sp->point[2];

=======
   oMinusC[2] = ray.point[2] - sp->point[2];   
  
>>>>>>> 4fabfe5fce695248b5da2724ded1529aa2c51cdb
   float _b = dot_product(ray.dir,oMinusC);
   float _c = dot_product(oMinusC,oMinusC) - (sp->rad * sp->rad);
   float det = _b * _b - _c;

   if (det < 0)
   {
      return -1;
   }

   float t0 = -_b - (float)sqrt(det);
   float t1 = -_b + (float)sqrt(det);

   if (t0 >= 0 && t1 >= 0.0)
   {
      if (t0 < t1)
      {
         return t0;
      }
      else if (t1 < t0)
      {
         return t1;
      }
   }
   else if (t0 >= 0)
   {
      return t0;
   }
   else if (t1 >= 0)
   {
      return t1;
   }

   return -1;
}

__global__ void Intersection_Test(ray_dev_t *rays, sphere_dev_t *spheres, intersection_t *intersects, int size, int width, int height)
{
   float t;
   bool hitFound = false;
   float curDepth = -1.0;
   int x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
   int y = blockIdx.y * BLOCK_SIZE + threadIdx.y;

   if(x > width || y > height)
<<<<<<< HEAD
      return;

   ray_dev_t *ray = rays + x * height + y;
=======
       return;

   ray_dev_t *ray = rays + x * height + y;   
>>>>>>> 4fabfe5fce695248b5da2724ded1529aa2c51cdb
   intersection_t *result = intersects + x * height + y;

   result->index = -1;

   for (int i = 0; i < size; i++)
   {
      sphere_dev_t *curObject = &spheres[i];
      t = -1.0;
<<<<<<< HEAD
      t = hit(*ray, curObject);
=======
      t = hit(*ray, &t, curObject);
>>>>>>> 4fabfe5fce695248b5da2724ded1529aa2c51cdb

      if (t >= 0)
      {
         if (t >= 0 && (!hitFound || (hitFound && t < curDepth)))
         {
            curDepth = t;
            result->index = i;
         }
      }
      hitFound |= (t >= 0 && curDepth > 0.0);
   }
<<<<<<< HEAD

=======
   
>>>>>>> 4fabfe5fce695248b5da2724ded1529aa2c51cdb
   //Add intersection
   result->t = curDepth;
}

