#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include "vector.h"

// includes, project
#include <cutil.h>

#include "Intersection_K.h"

__device__ float dot_product(float *v1, float *v2)
{
   return v1[0] * v2[0] + v1[1] * v2[1] + v1[2] * v2[2];
}

__device__ float det3(float **v1)
{
   return v1[0][0] * v1[1][1] * v1[2][2] + v1[0][1] * v1[1][2] * v1[2][0] + v1[0][2] * v1[1][0] * v1[2][1] -
      v1[0][2] * v1[1][1] * v1[2][0] - v1[0][0] * v1[1][2] * v1[2][1] - v1[0][1] * v1[1][0] * v1[2][2];
   ;
}

__device__ float hit(ray_dev_t ray, triangle_dev_t *tri);
{
   float result = -1;
   float bBeta, bGamma, bT;
   float detA;

   float A[3][3];
   A[0][0] = tri->c1[0]-tri->c2[0];
   A[0][1] = tri->c1[0]-tri->c3[0];
   A[0][2] = ray.dir[0];
   A[1][0] = tri->c1[1]-tri->c2[1];
   A[1][1] = tri->c1[1]-tri->c3[1];
   A[1][2] = ray.dir[1];
   A[2][0] = tri->c1[2]-tri->c2[2];
   A[2][1] = tri->c1[2]-tri->c3[2];
   A[2][2] = ray.dir[2];

   detA = det3(A);

   float baryT[3][3];
   baryT[0][0] = tri->c1[0]-tri->c2[0];
   baryT[0][1] = tri->c1[0]-tri->c3[0];
   baryT[0][2] = tri->c1[0]-ray.point[0];
   baryT[1][0] = tri->c1[1]-tri->c2[1];
   baryT[1][1] = tri->c1[1]-tri->c3[1];
   baryT[1][2] = tri->c1[1]-ray.point[1];
   baryT[2][0] = tri->c1[2]-tri->c2[2];
   baryT[2][1] = tri->c1[2]-tri->c3[2];
   baryT[2][2] = tri->c1[2]-ray.point[2];

   bT = det3(baryT) / detA;
   if (bT < 0)
   {
      result = 0;
   }
   else
   {
      float baryGamma[3][3];

      baryGamma[0][0] = tri->c1[0]-tri->c2[0];
      baryGamma[0][1] = tri->c1[0]-ray.point[0];
      baryGamma[0][2] = ray.dir[0];
      baryGamma[1][0] = tri->c1[1]-tri->c2[1];
      baryGamma[1][1] = tri->c1[1]-ray.point[1];
      baryGamma[1][2] = ray.dir[1];
      baryGamma[2][0] = tri->c1[2]-tri->c2[2];
      baryGamma[2][2] = tri->c1[2]-ray.point[2];
      baryGamma[2][3] = ray.dir[2];

      bGamma = det3(baryGamma) / detA;
      if (bGamma < 0 || bGamma > 1)
      {
         result = 0;
      }
      else
      {
         float baryBeta[3][3];

         baryBeta[0][0] = tri->c1[0]-ray.point[0];
         baryBeta[0][1] = tri->c1[0]-tri->c3[0];
         baryBeta[0][2] = ray.dir[0];
         baryBeta[1][0] = tri->c1[1]-ray.point[1];
         baryBeta[1][1] = tri->c1[1]-tri->c3[1];
         baryBeta[1][2] = ray.dir[1];
         baryBeta[2][0] = tri->c1[2]-ray.point[2];
         baryBeta[2][1] = tri->c1[2]-tri->c3[2];
         baryBeta[2][2] = ray.dir[2];

         bBeta = det3(baryBeta) / detA;
         if (bBeta < 0 || bBeta > 1 - bGamma)
         {
            result = 0;
         }
      }
   }

   if (result != 0)
   {
      result = bT;
   }
   return (result);
}

__device__ float hit(ray_dev_t ray, sphere_dev_t *sp)
{
   float oMinusC[3];
   oMinusC[0] = ray.point[0] - sp->point[0];
   oMinusC[1] = ray.point[1] - sp->point[1];
   oMinusC[2] = ray.point[2] - sp->point[2];   
  
   float _b = dot_product(ray.dir,oMinusC);
   float _c = dot_product(oMinusC,oMinusC) - (sp->rad * sp->rad);
   float det = _b * _b - _c;

   if (det < 0)
   {
      return -1;
   }

   float t0 = -_b - (float)sqrt(det);
   float t1 = -_b + (float)sqrt(det);

   if (t0 >= 0 && t1 >= 0.0)
   {
      if (t0 < t1)
      {
         return t0;
      }
      else if (t1 < t0)
      {
         return t1;
      }
   }
   else if (t0 >= 0)
   {
      return t0;
   }
   else if (t1 >= 0)
   {
      return t1;
   }

   return -1;
}

__global__ void Intersection_Test(ray_dev_t *rays, sphere_dev_t *spheres, intersection_t *intersects, int size, int width, int height)
{
   float t;
   bool hitFound = false;
   float curDepth = -1.0;
   int x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
   int y = blockIdx.y * BLOCK_SIZE + threadIdx.y;

   if(x > width || y > height)
      return;

   ray_dev_t *ray = rays + x * height + y;

   intersection_t *result = intersects + x * height + y;

   result->index = -1;

   for (int i = 0; i < size; i++)
   {
      sphere_dev_t *curObject = &spheres[i];
      t = -1.0;
      t = hit(*ray, curObject);

      if (t >= 0)
      {
         if (t >= 0 && (!hitFound || (hitFound && t < curDepth)))
         {
            curDepth = t;
            result->index = i;
         }
      }
      hitFound |= (t >= 0 && curDepth > 0.0);
   }

   //Add intersection
   result->t = curDepth;
}
