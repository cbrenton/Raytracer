#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cstdlib>
#include <string>
#include <string.h>
#include <sstream>
#include <sys/time.h>
#include <sys/ioctl.h>
#include <float.h>

#include "Pixel.h"
#include "Image.h"
#include "Scene.h"

#define POV_EXT ".pov"
#define DEFAULT_W 256
#define DEFAULT_H 256
#define AA_RAYS 8

// Set to 0 to turn off progress tracking.
#define SHOW_PROGRESS 1

// Determines the length of the progress bar. If your terminal is being overrun, try decreasing this.
#define BAR_LEN 20

using namespace std;

Image image;
Scene* scene;
string inputFileName;
string filename;
int width = DEFAULT_W;
int height = DEFAULT_H;

void setWidth(char* strIn)
{
   if (strIn[0] == '=')
   {
      width = atoi(strIn + 1);
   }
   else
   {
      width = atoi(strIn);
   }
   if (width <= 0)
   {
      cerr << "Invalid width.\n";
      exit(EXIT_FAILURE);
   }
}

void setHeight(char* strIn)
{
   if (strIn[0] == '=')
   {
      height = atoi(strIn + 1);
   }
   else
   {
      height = atoi(strIn);
   }
   if (height <= 0)
   {
      cerr << "Invalid height: " << height << endl;
      exit(EXIT_FAILURE);
   }
}

void setFilename(char* strIn)
{
   string name = "";
   if (strIn[0] == '=')
   {
      name = strIn[1];
   }
   else
   {
      name = strIn;
   }
   inputFileName = name;
   int extIndex = (int)name.rfind(POV_EXT);
   filename = name.substr(0, extIndex);
   filename.append(".tga");
}

// Testing triangle intersection. Should hit at 3.
void goodTest()
{
   vec3_t c1 = vec3_t(-1, -1, 3);
   vec3_t c2 = vec3_t(0, 1, 3);
   vec3_t c3 = vec3_t(1, -1, 3);
   Triangle *tri = new Triangle(c1, c2, c3);
   Ray *ray = new Ray(vec3_t(0, 0, 0), vec3_t(0, 0, 1));
   float t = -1.0f;
   bool hit = tri->hit(*ray, &t);
   if (hit)
   {
      std::cout << "hit at " << t << std::endl;
   }
   else
   {
      std::cout << "no hit: " << t << std::endl;
   }

   // Should not hit.
}

// Testing triangle intersection. Should not hit.
void badTest()
{
   vec3_t c1 = vec3_t(-1, 0.1f, 3);
   vec3_t c2 = vec3_t(0, 1, 3);
   vec3_t c3 = vec3_t(1, 0.1f, 3);
   Triangle *tri = new Triangle(c1, c2, c3);
   Ray *ray = new Ray(vec3_t(0, 0, 0), vec3_t(0, 0, 1));
   float t = -1.0f;
   bool hit = tri->hit(*ray, &t);
   if (hit)
   {
      std::cout << "hit at " << t << std::endl;
   }
   else
   {
      std::cout << "no hit: " << t << std::endl;
   }
}

float randFloat()
{
   float maxJitter = 0.5f;
   float minJitter = -0.5f;
   //float maxJitter = scene->camera->right.length() / (float)image.width;
   //float minJitter = scene->camera->up.length() / (float)image.height;
   return (float)rand() / (float)FLT_MAX * (maxJitter - minJitter) - minJitter;
}

void printProgress(struct timeval startTime, int d, int total, int freq)
{
   // Initialize timekeeping variables.
   float timeLeft;
   float dt = 0;
   int seconds, useconds;
   int min, sec, ms;
   int dMin, dSec, dMs;
   min = sec = ms = dMin = dSec = dMs = 0;

   // Set padding for strings to their length (minus one for null
   // terminating character) plus a specified value.
   int strPad = 3;
   int pad = 4;

   // Get terminal width.
   struct winsize w;
   ioctl(0, TIOCGWINSZ, &w);
   int termW = w.ws_col;

   // Length of time string.
   int timeLen = 8;
   // Length of percent string.
   int percentLen = 7;

   int maxBarLen = (pad * 2 + strPad * 2) + (strlen("elapsed:") - 1)
      + (strlen("eta:") - 1) + (timeLen * 2) + (percentLen + 1)
      + (BAR_LEN + 2) + 1;
   int midBarLen = (pad + strlen("eta:") - 1 + strPad + timeLen
         + (percentLen + 1) + 1);
   int minBarLen = percentLen + 1;

   //bool fullProgressEnabled = maxBarLen > BAR_LEN;
   bool fullProgressEnabled = maxBarLen < termW;
   bool midProgressEnabled = midBarLen < termW;
   bool minProgressEnabled = minBarLen < termW;

   if (d % freq == 0 || d == total - 1)
   {
      // Get time.
      struct timeval curTime;
      gettimeofday(&curTime, NULL);
      seconds = (int)curTime.tv_sec - startTime.tv_sec;
      useconds = (int)curTime.tv_usec - startTime.tv_usec;
      dt = (float)(((seconds) * 1000 + useconds/1000.0) + 0.5);
      float percent = (float)(d + 1) / (float)total;

      timeLeft = ((float)dt / percent - (float)dt) / 1000.0f;

      // Calculate time data;
      min = (int)timeLeft / 60;
      sec = (int)timeLeft % 60;
      ms = (int)(timeLeft * 100) % 60;

      dMin = (int)(dt / 1000) / 60;
      dSec = (int)(dt / 1000) % 60;
      dMs = (int)(dt / 10) % 60;

      if (fullProgressEnabled)
      {
         // Print everything.
         std::string progress;
         // Fill progress bar.
         progress += "[";
         for (int j = 0; j < BAR_LEN; j++)
         {
            float j_percent = (float)j / (float)BAR_LEN;
            if (j_percent <= percent)
            {
               progress += "=";
            }
            else
            {
               progress += "-";
            }
         }
         progress += "]";

         // Print data.
         printf("\r%s%*s%02d:%02d:%02d",
               "elapsed:", strPad, "", dMin, dSec, dMs);
         printf("%*s%s%*s%02d:%02d:%02d",
               pad, "", "eta:", strPad, "", min, sec, ms);
         // Display progress bar.
         printf("%*s%*.2f%% %s",
               pad, "", percentLen - 2, percent * 100.0f, progress.c_str());
      }
      else if (midProgressEnabled)
      {
         // Print the percent and the ETA.
         printf("\r%-*s %02d:%02d:%02d",
               strlen("eta:") - 1 + strPad, "eta:", min, sec, ms);
         printf("%*s%.2f%%",
               pad, "", percent * 100.0f);
      }
      else if (minProgressEnabled)
      {
         // Print only the percent.
         printf("\r%.2f%%", percent * 100.0f);
      }

      /*
         else
         {
         printf("Warning: terminal must be at least %d characters wide. Data will not be displayed.\n", minBarLen);
         }
         printf("terminal width: %d (%d)", termW, maxBarLen);
         */

      // Flush stdout to print stats.
      fflush(stdout);
   }
}

int main(int argc, char **argv)
{
   /*
   // Triangle testing.
   goodTest();
   badTest();
   exit(0);
   */

   srand(time(NULL));

   int c;
   while ((c = getopt(argc, argv, "w:W:h:H:i:I:")) != -1)
   {
      switch (c)
      {
      case 'w': case 'W':
         setWidth(optarg);
         break;
      case 'h': case 'H':
         setHeight(optarg);
         break;
      case 'i': case 'I':
         setFilename(optarg);
         break;
      default:
         cerr << "Invalid command-line argument -" << c << endl;
         exit(EXIT_FAILURE);
         break;
      }
   }
   for (int i = 1; i < argc; i++)
   {
      if (argv[i][0] == '+')
      {
         switch(argv[i][1])
         {
         case 'w': case 'W':
            setWidth(argv[i] + 2);
            break;
         case 'h': case 'H':
            setHeight(argv[i] + 2);
            break;
         case 'i': case 'I':
            setFilename(argv[i] + 2);
            break;
         default:
            cerr << "Invalid command-line argument -" << c << endl;
            exit(EXIT_FAILURE);
            break;
         }
      }
   }

   ifstream inputFileStream(inputFileName.c_str());

   scene = Scene::read(inputFileStream);

   image = Image(width, height);
   image.filename = filename;
   image.init();

   // Make AA_RAYS subpixels per pixel.
   Ray aRayArray [width][height];

   float l = -scene->camera->right.length() / 2;
   float r = scene->camera->right.length() / 2;
   float b = -scene->camera->up.length() / 2;
   float t = scene->camera->up.length() / 2;

   for (int i = 0; i < image.width; i++)
   {
      for (int j = 0; j < image.height; j++)
      {
         //Ray *curRay = new Ray();
         //for (int k = 0; k < AA_RAYS; k++)
         //{
         //float xJitter = randFloat();
         //float yJitter = randFloat();
         float xJitter = 0.0f;
         float yJitter = 0.0f;
         float uScale = (float)(l + (r - l) * (((float)i + xJitter + 0.5f)
                  / (float)image.width));
         //float uScale = (float)(l + (r - l) * (((float)i + 0.5f)
         float vScale = (float)(b + (t - b) * (((float)j + yJitter + 0.5f)
                  / (float)image.height));
         //float vScale = (float)(b + (t - b) * (((float)j + 0.5f)
         float wScale = -1;
         vec3_t sVector = scene->camera->location;
         vec3_t uVector = scene->camera->right;
         vec3_t vVector = scene->camera->up;
         vec3_t wVector = scene->camera->look_at - scene->camera->location;
         uVector.normalize();
         vVector.normalize();
         wVector.normalize();
         // Left-handed.
         wVector *= -1;
         uVector *= uScale;
         vVector *= vScale;
         wVector *= wScale;
         sVector += uVector;
         sVector += vVector;
         sVector += wVector;
         vec3_t rayDir = uVector + vVector + wVector;
         rayDir.normalize();
         vec3_t curPoint = vec3_t(scene->camera->location);
         //Ray *curRay = new Ray(curRay->point, curRay->dir);
         //curRay = new Ray(curRay->point, curRay->dir);
         Ray *curRay = new Ray(curPoint, rayDir);
         aRayArray[i][j] = *curRay;
         /*
            if (k == 0)
            {
            aRayArray[i][j] = *curRay;
            }
            else
            {
         //aRayArray[i][j].dirs[k] = curRay->dir;
         //aRayArray[i][j].point = curRay->point;
         }
         */
         delete curRay;
      }
      //}
   }

   // Initialize variables for timekeeping.
   struct timeval startTime;
   gettimeofday(&startTime, NULL);

   std::cout << "starting intersection tests." << std::endl;
   for (int i = 0; i < image.width; i++)
   {
      for (int j = 0; j < image.height; j++)
      {
         Pixel *result = new Pixel();
         for (int k = 0; k < AA_RAYS; k++)
            //for (int k = 0; k < 1; k++)
         {
            // Find intersections and get pixel data.
            //Ray *tmpRay = new Ray(aRayArray[i][j].point, aRayArray[i][j].dirs[k]);
            /*
               Ray tmpRay = aRayArray[i][j];
               HitData *data = scene->getIntersect(tmpRay);
               if (data->hit)
               {
            //result->add(scene->seekLight(data, aRayArray[i][j].dirs[k]));
            //result->add(scene->seekLight(data, tmpRay.dir));
            result->add(scene->seekLight(data, aRayArray[i][j].dir));
            }
            */
            HitData *data = scene->getIntersect(aRayArray[i][j]);
            if (data->hit)
            {
               result->add(scene->seekLight(data, aRayArray[i][j].dir));
            }
            delete data;
         }
         result->multiply(1.0f / (float)AA_RAYS);
         if (SHOW_PROGRESS)
         {
            // Set the frequency of ticks to update every .01%, if possible.
            int tick = std::max(image.width*image.height/AA_RAYS / 10000, 1);
            printProgress(startTime, i * image.height + j,
                  image.width * image.height, tick);
         }
         image.setPixel(i, j, result);
         delete result;
      }
   }

   if (SHOW_PROGRESS)
   {
      std::cout << std::endl;
   }

   // Write image to file.
   image.write();

   return 0;
}
